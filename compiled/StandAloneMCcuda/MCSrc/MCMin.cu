#include "hip/hip_runtime.h"
/**************************************************************
 * \file MCMin.cu
 *
 * \brief CUDA implementation of Monte Carlo Minimization
 * \author M.P. Kuchera and Y. Ayyad
 * \date created 19 March 2016
 *
 **************************************************************/



#include "MCMin.cuh"

#include <iostream>
#include <algorithm>
#include "TRandom.h"
#include "TH1F.h"
#ifdef _OPENMP
#include <omp.h>
#endif
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#define cRED "\033[1;31m"
#define cYELLOW "\033[1;33m"
#define cNORMAL "\033[0m"
#define cGREEN "\033[1;32m"

#define ITER 400
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
//Global variables on the GPU
__device__ Double_t sm1;
__device__ Double_t m;
__device__ Double_t dzstep;
__device__ Int_t    integrationsteps;
__device__ Double_t restmass;
__device__ Double_t esm;
__device__ Double_t iz1;
__device__ Double_t z1;
__device__ Double_t B0;
//__device__ Double_t B;
__device__ Double_t fZk;
__device__ Double_t phimin[ITER];//=phi0;
__device__ Double_t thetamin[ITER];//=theta0;
__device__ Double_t bromin[ITER];//=bro;
__device__ Double_t Bmin[ITER];
//__device__ Double_t phi0;
//__device__ Double_t theta0;
//__device__ Double_t bro;
  /////////// Initial parameters///////////////
__device__ Double_t xmin;//= parameter[0]/10.0; //  ! at ztb=394 in cm
__device__ Double_t ymin;//= parameter[1]/10.0;
__device__ Double_t zmin;//= fZk/10.0 - (fEntTB-parameter[3])*dzstep;  //Micromegas Origin  at 100 cm of the entrance
__device__ Double_t TBmin;// = parameter[3]*dzstep;
__device__ Double_t fEntTB;
__device__ Double_t phi0[ITER];
__device__ Double_t bro[400];
__device__ Double_t theta0[400];
__device__ Double_t B[400];
__device__ Double_t brotheta[400];


__shared__ Double_t fThetaPad;
__shared__ Double_t fThetaTilt;
__shared__ Double_t fThetaLorentz;
__shared__ Double_t fThetaRot;

//_device__ Double_t e0sm[400];
//__device__  Double_t phi0=TMath::Pi()-parameter[4]-115*TMath::Pi()/180.0;
  
//__device__  Double_t bro=parameter[5]*B0/1000.0;// !Tm*/
//__device__  Double_t theta0=parameter[6];
  ////////////////////////////////////////////

typedef struct {
  Double_t X;
  Double_t Y;
  Double_t Z;
} vec3;

__global__ void setGlobalGPUVariables(){
  m                  = 1;
  sm1                = m;
  dzstep             = 5.20*80.0/1000.0;//  !unit cm
  integrationsteps   = 10;
  iz1                = 1;
  z1                 = iz1;
  restmass           = sm1*931.49432;
  esm                = z1*1.75879e-3*0.510998918/restmass;// ![e/m electron cm**2/(Volt*nsec**2] this is not the energy/mass but charge/mass
  B0                 = 1.66;  // !	magnetic field
  // B                  = B0*10000.; // !conversion of T en Gauss
  fZk              = 1000.0;
  fEntTB           = 280;
  // B                = B0*10000;
  fThetaLorentz    = -6.6*HIP_PI_F/180.0;
  fThetaRot        = -13.0*HIP_PI_F/180.0;
  fThetaTilt       = 7.4*HIP_PI_F/180.0;
  fThetaPad        = 113.7*HIP_PI_F/180.0;
}
__device__ vec3 setXYZ(Double_t x, Double_t y, Double_t z){
  vec3 vec;
  vec.X = x;
  vec.Y = y;
  vec.Z = z;
  return vec;
}
__device__ vec3 TransformIniPos(Double_t x,Double_t y, Double_t z)
{

   vec3 PosIniCmm;

   Double_t x_det = x*cos(fThetaPad)  + y*sin(fThetaPad);
   Double_t y_det = -x*sin(fThetaPad) + y*cos(fThetaPad);
   Double_t z_det = z;

   Double_t x_sol = x_det;
   Double_t z_sol = ( z_det*cos(fThetaTilt) + y_det*sin(fThetaTilt) + fZk/10.0*pow(sin(fThetaTilt),2.0) ) ;
   Double_t y_sol = ( y_det + (fZk/10.0-z_sol)*sin(fThetaTilt) )/ cos(fThetaTilt);

   Double_t z_cmm = z_sol;
   Double_t x_cmm = x_sol + z_cmm*sin(fThetaLorentz)*sin(fThetaRot);
   Double_t y_cmm = y_sol - z_cmm*sin(fThetaLorentz)*cos(fThetaRot);

   PosIniCmm = setXYZ(x_cmm,y_cmm,z_cmm);

   return PosIniCmm;


}
__device__ __host__ void MCMinimization::GetEnergy(Double_t M,Double_t IZ,Double_t BRO,Double_t &E) {
  
}

__device__ Double_t GetEnergy(Double_t M,Double_t IZ,Double_t BRO) {
  Float_t  AM=931.5;
  Float_t X=BRO/0.1439*IZ/M;
  X=powf(X,2.0);
  X=2.*AM*X;
  X=X+powf(AM,2.0);
  Double_t E=sqrt(X)-AM;
  return E;
}



__global__ void setInitialParameters(Double_t* parameter){
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if(idx < 400){
  
    phimin[idx]=phi0[idx];
    thetamin[idx]=theta0[idx];
    bromin[idx]=bro[idx];
    
    Double_t tm=0.0;
    //Int_t iteration=0;

    /////////// Initial parameters///////////////
    xmin= parameter[0]/10.0; //  ! at ztb=394 in cm
    ymin= parameter[1]/10.0;
    zmin= fZk/10.0 - (fEntTB-parameter[3])*dzstep;  //Micromegas Origin  at 100 cm of the entrance
    TBmin = parameter[3]*dzstep;
    phi0[idx]=HIP_PI_F-parameter[4]-115*HIP_PI_F/180.0;
    B[idx]                  = B0*10000.; // !conversion of T en Gauss
    bro[idx]=parameter[5]*B0/1000.0;// !Tm*/
    theta0[idx]=parameter[6];
    brotheta[idx] = bromin[idx]/sin(thetamin[idx]);
      
    fThetaPad = 0;
    fThetaTilt = 0;
    fThetaLorentz = 0;
    fThetaRot = 0;
    
    //  brotheta[idx]=bromin[idx]/sin(thetamin[idx]);
    Double_t e0sm = GetEnergy(sm1,z1,brotheta[idx]);
    printf(" Energy of the proton : %f MeV\n",e0sm);
    Double_t chimininit=1.e6;
    if(bro==0 || isnan(e0sm) || e0sm>100.0){
      printf(" Invalid energy !\n\n");//<<cNORMAL<<std::endl;
      return;// kFALSE;
  }

  ////////////////////////////////////////////
  }
  if(idx==0){
    printf(cGREEN "\n\n ============================\n");
    printf(" Starting Monte Carlo event \n");
    printf(" X : %f cm  - Y : %f cm - Z : %f cm \n",xmin,ymin,zmin);//<<std::endl;
    printf(" Brho : %f Tm \n",bro[idx]);//<<std::endl;
    printf(" Radius of curvature : %f mm\n",parameter[5]);//<<" mm "<<std::endl;
    printf(" Scattering Angle : %f deg\n",theta0[idx]*180.0/HIP_PI_F);//<<" deg "<<std::endl;
    printf(" Azimutal Angle : %f deg\n",phi0[idx]*180.0/HIP_PI_F);
    printf(" Length of the experimental data : %f\n============================\n" cNORMAL,parameter[7]);//<<cNORMAL<<std::endl;
  }
  
}
__global__ void init_stuff(hiprandState* state)
{
int idx = blockIdx.x*blockDim.x+threadIdx.x;
hiprand_init(1337,idx,0,&state[idx]);
}

__global__ void make_rand(hiprandState* state,float* randArray)
{
int idx = blockIdx.x*blockDim.x+threadIdx.x;
randArray[idx]=hiprand_uniform(&state[idx]);
}


__global__ void calcInner(int i, hiprandState* state, Double_t* parameter) {
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  // printf(cRED "idx = %d\n" cNORMAL,idx);
  if(idx < 400) {
     Double_t xcmm = 0;
     Double_t ycmm = 0;
     Double_t zcmm = 0;
     
     Double_t xsol = 0;
     Double_t ysol = 0;
     Double_t zsol = 0;
     
     Double_t xdet = 0;
     Double_t ydet = 0;
     Double_t zdet = 0;
     
     Double_t xpad = 0;
     Double_t ypad = 0;
     Double_t zpad = 0;
     
     Double_t xTBCorr[100];
     Double_t yTBCorr[100];
     Double_t zTBCorr[100];
     
    //setInitialParameters();
    Float_t factstep=1.0/(pow(1.4,(double)i));
    Float_t step1=2*factstep;// !theta in deg
    Float_t step2=2*factstep;//  !phi in deg
    Float_t step3=0.2*factstep;//! broradius in realtive vaue
    Float_t step4=0.3*factstep;//!x0 in cm
    Float_t step5=0.3*factstep;// !y0
    Float_t step6=0.5*factstep;//!z0
    Float_t step7=0.0*factstep;//B
    Float_t step8=0.0*factstep;//Density
    Double_t e0sm            = 0.0;
    Double_t tm=0.0;


    Bmin[idx] = B[idx];
    bro[idx]=bromin[idx]*(1.+(0.5-hiprand_uniform(&state[idx]))*step3); //!in Tm
    theta0[idx]=thetamin[idx]+step1*(0.5-hiprand_uniform(&state[idx]))*0.01745;
    brotheta[idx]=bro[idx]/sin(theta0[idx]);//  !initial bro corrected for angle
    
    phi0[idx]= phimin[idx]+ step2*(0.5-hiprand_uniform(&state[idx]))*0.01745;
    B[idx]=Bmin[idx]*(1.+step7*(0.5-hiprand_uniform(&state[idx])));
    e0sm = GetEnergy(sm1,z1,brotheta[idx]);
    Double_t e0ll=e0sm*sm1;
    Double_t e0=e0ll*1000000.;// !conversion from MeV in eV kinetic energy
                              //	esm= 1.75879e-3
    Double_t beta2=2.*e0ll/(sm1*931.49);
    Double_t ekin=e0ll;
    Double_t beta0=sqrt(beta2);//    ![cm/nsec] 

    Double_t ecinsm=ekin/sm1;
    Double_t eloss=ekin;
    //	dedx=s*dens  !de/dx in [MeV/cm] !only defined after 1st tour
    //          dzstep=0.221  !unit cm
    //	boucle d'integration
    Int_t  ipr=0;
    Double_t  range=0.0;
    
    //        define initial conditions
    
    //         Transform initial parameters into lab
    Double_t  x = xmin + step4*(hiprand_uniform(&state[idx])-0.5); // ! ztb=394
    Double_t  y = ymin + step5*(hiprand_uniform(&state[idx])-0.5);
    Double_t  z = zmin + step6*(hiprand_uniform(&state[idx])-0.5);
    Double_t x_buff =x;
    Double_t y_buff =y;
    Double_t z_buff =z;  
    vec3 PosIniCmm;
    // Working here Mich //////TVector3 PosIniCmm = TransformIniPos(x,y,z); //Arguments in cm
    x=PosIniCmm.X;
    y=PosIniCmm.Y;
    z=PosIniCmm.Z;
    Double_t zmin_trans = z;
    Double_t  x0=x;
    Double_t  y0=y;
    Double_t  z0=z;
    
    Double_t v0=beta0*29.9792;//  !v in cm/ns
    Double_t dt=dzstep/(v0*cos(theta0[idx]));//![unite temps ns]
    dt=dt/(Float_t)integrationsteps; // NEW
    Double_t t=-dt;
    
    //           initial velocity vector
    //	v0=sqrt(v2)    ![cm/nsec]
    Double_t dxdt=v0*sin(theta0[idx])*cos(phi0[idx]);
    Double_t dydt=v0*sin(theta0[idx])*sin(phi0[idx]);
    Double_t dzdt=v0*cos(theta0[idx]);
    Double_t iprinttr=1000;
    Double_t dens=0.06363*18*(1+step8*(hiprand_uniform(&state[idx])-0.5))/20.;//  !DENSITY ISOBUTANE AT 20 TORR corrected 18 torr
    Double_t iterationmax=10000;
    ipr=0;
    
    //******************************************************************
    
    Int_t iteration=0;
    Int_t iterd=0;
    Int_t iterCorr=0;
    Int_t iterCorr_0=0; //offset correction for TB
    Int_t iterd0=0;
    Int_t iterCorrNorm=0;
    
    Int_t icnb;
    Int_t num_MC_Point = 0;
    
    for(Int_t k=0;k<iterationmax;k++)
      {	
	iteration++;
	Float_t factq=1.0;
	//iterd=(Int_t) k/integrationsteps;//NEW
	iterd=k;	
	
	xcmm = x*10.0;
	ycmm = y*10.0;
	zcmm = z*10.0;
	
	zcmm = -zcmm + 2*zmin_trans*10.0;
	xsol=xcmm-zcmm*sin(fThetaLorentz)*sin(fThetaRot);
	ysol=ycmm+zcmm*sin(fThetaLorentz)*cos(fThetaRot);
	zsol=zcmm;
	
	xdet = xsol;
	ydet = -(fZk-zsol)*sin(fThetaTilt) + ysol*cos(fThetaTilt);
	zdet = zsol*cos(fThetaTilt) - ysol*sin(fThetaTilt);
	
	xpad = xdet*cos(fThetaPad) - ydet*sin(fThetaPad);
	ypad = xdet*sin(fThetaPad) + ydet*cos(fThetaPad);
	zpad = zdet;

	iterCorr = (Int_t) (zpad/(dzstep*10) + 0.5);
	if(k==0) iterCorr_0 = iterCorr; //Offset renomarlization
	//std::cout<<" iterCorr : "<<iterCorr_0-iterCorr<<" iterd : "<<iterd<<std::endl;
	
	iterCorrNorm = iterCorr_0-iterCorr;
	if(iterCorrNorm<0) break;
	
	xTBCorr[iterCorrNorm] = xpad;
	yTBCorr[iterCorrNorm] = ypad;
	zTBCorr[iterCorrNorm] = zpad;
	
	//  if(iterCorrNorm!=icnb){
	//	xiter.push_back(xTBCorr[iterCorrNorm]);
	//yiter.push_back(yTBCorr[iterCorrNorm]);
	//ziter.push_back(zTBCorr[iterCorrNorm]);
	//    }
	icnb=iterCorrNorm;

	t=t+dt;
	Double_t ddxddt=esm*B[idx]*10.*dydt*factq;//  !remember esm =charge/masse
	Double_t ddyddt=-esm*(B[idx]*10.*dxdt)*factq;
	Double_t ddzddt=0.;
	x=x + dxdt*dt + 0.5*ddxddt*pow(dt,2.0);
	y=y + dydt*dt + 0.5*ddyddt*pow(dt,2.0);
	z=z + dzdt*dt + 0.5*ddzddt*pow(dt,2.0);
	dxdt=dxdt+ddxddt*dt;
	dydt=dydt+ddyddt*dt;
	dzdt=dzdt+ddzddt*dt;

	Double_t help=pow((dxdt*dt + 0.5*ddxddt*pow(dt,2.0)),2.0 );
	help=help+pow((dydt*dt + 0.5*ddyddt*pow(dt,2.0)),2.0);
	help=help+pow((dzdt*dt + 0.5*ddzddt*pow(dt,2.0)),2.0);
	help=sqrt(help);
	range=range+help;
	//std::cout<<" Range : "<<range<<std::endl;
	Double_t sloss = 0.0;
	
	Double_t  c0;
	 
	if(iz1==1){
	  c0=ekin;
	  if(m==2) c0=c0/2.0;
	  sloss=6.98*(1./pow(c0,0.83))*(1./(20.+1.6/pow(c0,1.3)))+0.2*exp(-30.*pow((c0-0.1),2.0)); //Old expression
	  //sloss = 0.3*TMath::Power((1./c0),0.78)*(1./(1.+0.023/TMath::Power(c0,1.37)));
	}
	if(iz1==6){
	  c0=ekin/6.;
	  sloss=36.*(1./pow(c0,0.83))*(1./(1.6+1.6/pow(c0,1.5)))+1.*exp(pow(-(c0-0.5),2.0));
	}
	if(iz1==2){
	  c0=ekin;
	  sloss=11.95*(1./pow(c0,0.83))*(1./(2.5+1.6/pow(c0,1.5)))+ 0.05*exp(pow(-(c0-0.5),2.0));
	}
	
	sloss= sloss*dens*help; //!energy loss with density and step
	Double_t vcin=sqrt(pow(dxdt,2.0)+pow(dydt,2.0)+pow(dzdt,2.0));// !v in cm/ns
	Double_t vsc=vcin/29.979; // !v/c
	Double_t beta=vsc;
	//ecinsm=931.494/2.*(vsc)**2*1./sqrt(1.-vsc**2) !relativistic
	Double_t ekindo=sm1*931.494*0.5*pow(vsc,2.0); //!nonrelativistic
	help=ekin;
	ekin=ekin-sloss;// !energy loss
	//help=ekin/help
	help=ekin/ekindo;
	help=sqrt(help);
	Double_t help1=help;
	ekindo=ekindo-sloss;
	dxdt=dxdt*help; // ! this is the introduction energy loss!!!!
	dydt=dydt*help;
	dzdt=dzdt*help;
	//dt=dzstep/(dzdt);
	dt=dzstep/(dzdt)/(Float_t) integrationsteps;//NEW
	//std::cout<<" z :"<<z<<" dt : "<<dt<<" dzstep : "<<dzstep<<" dzdt : "<<dzdt<<std::endl;
	//test if still in detector supposing radius of 25cm and lenght 100cm
	Double_t radp2=pow(x,2.0)+pow(y,2.0);
	Double_t radp=sqrt(radp2);
	//std::cout<<cRED<<" dt : "<<dt<<cNORMAL<<std::endl;
	
	//  if(radp>25.0) break;//  !this limits the radial distance of the trajectories taken into ccount
	//	if(z.gt.ztot) go to 100
	if(zTBCorr[iterCorrNorm]<0.0) break;
                                                                //std::cout<<" Ekin : "<<ekin<<std::endl;
	if(ekin<0.01 || isnan(ekin)) break; 
	//if(ekin<0) std::cout<<" Ekin "<<std::endl;	
	
      } // k loop spiral integration
    tm=tm+t;
    Int_t iterh = (Int_t)(iteration/integrationsteps);
    
    Int_t imaxchi2=max(iterCorrNorm,(Int_t) parameter[7]); //NEW
    //if(true){  
  
    if(idx==0){
      printf(cRED " Imaxchi2 : %d \n",imaxchi2);//<<std::endl;
      printf(" iterCorrNorm : %d\n", iterCorrNorm);//<<std::endl;
      printf(" iterh : %d\n",iterh);//<<std::endl;
      printf(" parameter[7] : %f\n" cNORMAL,parameter[7]);//<<std::endl;
      //printf(" Num of interpolated exp points : %d" cNORMAL,numIntPoints);//<<cNORMAL<<std::endl;
    }
      //std::cout<<" iterCorrNorm : "<<iterCorrNorm<<std::endl;
      //std::cout<<" iterh : "<<iterh<<std::endl;
      //std::cout<<" parameter[7] : "<<parameter[7]<<std::endl;
      //std::cout<<" Num of interpolated exp points : "<<numIntPoints<<cNORMAL<<std::endl;
      //}
  }
}

MCMinimization::MCMinimization()
{
  setGlobalGPUVariables<<<1,1>>>();
  fThetaMin=0.0;
  fEnerMin=0.0;
  fBrhoMin=0.0;
  fBMin=0.0;
  fPhiMin=0.0;
  fDensMin=0.0;
  fVertexEner=0.0;

  fThetaLorentz    = -6.6*TMath::Pi()/180.0;
  fThetaRot        = -13.0*TMath::Pi()/180.0;
  fThetaTilt       = 7.4*TMath::Pi()/180.0;
  fThetaPad        = 113.7*TMath::Pi()/180.0;
  fEntTB           = 280;
  fZk              = 1000.0;

}

MCMinimization::~MCMinimization()
{

}

void MCMinimization::ResetParameters()
{
  FitParameters.sThetaMin    = 0;
  FitParameters.sEnerMin     = 0;
  FitParameters.sPosMin.SetXYZ(0,0,0);
  FitParameters.sBrhoMin     = 0;
  FitParameters.sBMin        = 0;
  FitParameters.sPhiMin      = 0;
  FitParameters.sChi2Min     = 0;
  FitParameters.sVertexPos.SetXYZ(0,0,0);
  FitParameters.sVertexEner  = 0;
  FitParameters.sMinDistAppr = 0;
  FitParameters.sNumMCPoint  = 0;
  FitParameters.sNormChi2    = 0;

}
Bool_t MCMinimization::MinimizeOpt(Double_t* parameter,ATEvent *event){
  hipSetDevice(0);
  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hiprandState* d_state;
  Double_t* d_params;
  //int nThreads = 400;
  int nThreads = 400;
  int nBlocks = 1;
  hipMalloc(&d_state,nThreads*nBlocks);
  hipMalloc(&d_params,sizeof(Double_t)*8);
  hipMemcpy(d_params,parameter,sizeof(Double_t)*8,hipMemcpyHostToDevice);
  init_stuff<<<nBlocks,nThreads>>>(d_state);
  //make_rand<<<nblocks,nthreads>>>(d_state,randArray);
  setInitialParameters<<<1,1>>>(d_params);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
  //std::cout << "Initial parameters set\n";
  /////////// Initial parameters///////////////
  // Double_t xmin= parameter[0]/10.0; //  ! at ztb=394 in cm
  // Double_t ymin= parameter[1]/10.0;
  // Double_t zmin= fZk/10.0 - (fEntTB-parameter[3])*dzstep;  //Micromegas Origin  at 100 cm of the entrance
  // Double_t TBmin = parameter[3]*dzstep;
  // Double_t phi0=TMath::Pi()-parameter[4]-115*TMath::Pi()/180.0;
  
  // Double_t bro=parameter[5]*B0/1000.0;// !Tm*/
  // Double_t theta0=parameter[6];
  // ////////////////////////////////////////////
  
  // Double_t phimin=phi0;
  // Double_t thetamin=theta0;
  // Double_t bromin=bro;
  // Double_t brotheta=bromin/TMath::Sin(thetamin);
  // Double_t tm=0.0;
  // Int_t iteration=0;
  
  kVerbose = kTRUE;
  kDebug   = kFALSE;
  
  for(int i=0;i<5;i++){
    calcInner<<<nBlocks,nThreads>>>(0,d_state,d_params);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk(hipDeviceSynchronize());
  }
  hipFree(d_state);
  hipFree(d_params);
}

std::vector<ATHit> MCMinimization::GetTBHitArray(Int_t TB,std::vector<ATHit> *harray)
{

     
}

/*void MCMinimization::GetEnergy(Double_t M,Double_t IZ,Double_t BRO,Double_t &E){


  }*/
TVector3 MCMinimization::TransformIniPos(Double_t x,Double_t y, Double_t z)
{
}

TVector3 MCMinimization::InvTransIniPos(Double_t x,Double_t y, Double_t z)
{
}
void MCMinimization::BackwardExtrapolation()
{
}
